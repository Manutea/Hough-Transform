#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "thrust/device_vector.h"
#include "thrust/host_vector.h"
#include <stdio.h>
#include "Bitmap_image.h"

#define DEG2RAD (3.1415926535f/180.0f)

/** @struct line
* @brief list possible lines.
* @details Used to store data to retrace a line.
*/
struct line
{
    int x1, y1, x2, y2;
    bool isAline = false;
};

const thrust::host_vector<rgb_t> getPixels(const bitmap_image&, const int, const int);
void drawResult(const int, const int, const std::string&, const int, const line*, const rgb_t*);
__global__ void cudaAccumulator(const int, const int, const int, const double, const double, const double, const rgb_t*, int*);
__global__ void cudaGetLines(const int, const int, const int, const int, const int, const int*, line*);

/**
* @brief main.
*
* @param[in] argv[1], the input image path.
* @param[in] argv[2], the output result image path.
*/
int main(int argc, char* argv[])
{
    const std::string inputImagePath = argv[1];
    const std::string outputImagePath = argv[2];

    bitmap_image image(inputImagePath);
    if (!image)
    {
        printf("Error - Failed to open: input.bmp\n");
        return 1;
    }

    const auto theshold = 250;
    const auto height = image.height();
    const auto width  = image.width();

    const thrust::host_vector<rgb_t> h_pixels = getPixels(image, width, height);

    const auto houghH   = ((sqrt(2.0) * (height > width ? height : width)) / 2.0);
    const auto centerX  = width / 2.0;
    const auto centerY  = height / 2.0;
    const auto accuH = (int)(houghH * 2.0);
    const auto accuW = 180;
    const auto accuSize = accuH * accuW;

    const auto threadX = 8;
    const auto threadY = 8;

    const thrust::device_vector<rgb_t> d_pixels = h_pixels;
    const rgb_t* pixelsBufferArray = thrust::raw_pointer_cast(&d_pixels[0]);

    thrust::device_vector<int> d_accu(accuSize, 0);
    auto* accuBufferArray = thrust::raw_pointer_cast(&d_accu[0]);

    const dim3 accBlocks(width / threadX + 1, height / threadY + 1);
    const dim3 accThreads(threadX, threadY);

    cudaAccumulator << <accBlocks, accThreads >> > (theshold, width, height, centerX, centerY, houghH, pixelsBufferArray, accuBufferArray);
    hipDeviceSynchronize();

    const dim3 linesBlocks(accuW / threadX + 1, accuH / threadY + 1);
    const dim3 linesThreads(threadX, threadY);

    thrust::device_vector<line> d_lines(accuSize);
    line* linesBufferArray = thrust::raw_pointer_cast(&d_lines[0]);
    cudaGetLines << <linesBlocks, linesThreads >> > (theshold, accuW, accuH, width, height, accuBufferArray, linesBufferArray);
    hipDeviceSynchronize();

    const thrust::host_vector<line> h_lines = d_lines;

    drawResult(width, height, outputImagePath, accuSize, h_lines.data(), h_pixels.data());

    return 0;
}

/**
* @brief Get the pixels value from the image.
* 
* @param[in] image is a reference of te bitmap_image variable.
* @param[in] width, the width of the image to analyze.
* @param[in] height, the height of the image to analyze.
* 
* @returns a host vector contanining the rgb values.
*/
const thrust::host_vector<rgb_t> getPixels(const bitmap_image& _image, const int _width, const int _height) {
    const auto size = _height * _width;
    thrust::host_vector<rgb_t> pixels(size);

    for (auto y = 0; y < _height; ++y) {
        for (auto x = 0; x < _width; ++x) {
            const auto index = _width * y + x;
            _image.get_pixel(x, y, pixels[index]);
        }
    }

    return pixels;
}

/**
* @brief draw Hough lines in a image file.
*
* @param[in] width, the width of the image to analyze.
* @param[in] height, the height of the image to analyze.
* @param[in] str, output direction to write the image.
* @param[in] linesSize, the array size of the lines.
* @param[in] lines, the lines to draw.
* @param[in] pixels, array contanining the rgb values from the image to analyse.
*/
void drawResult(const int _width, const int _height, const std::string& _str, const int _linesSize, const line* _lines, const rgb_t* _pixels) {
    bitmap_image imageLines(_width, _height);
    image_drawer draw(imageLines);

    for (auto y = 0; y < _height; ++y) {
        for (auto x = 0; x < _width; ++x) {
            const auto index = _width * y + x;
            const rgb_t pixel = _pixels[index];
            imageLines.set_pixel(x, y, pixel);
        }
    }

    draw.pen_color(255, 0, 0);

    for (auto i = 0; i < _linesSize; ++i) {
        const auto line = _lines[i];
        if (line.isAline)
            draw.line_segment(line.x1, line.y1, line.x2, line.y2);
    }

    imageLines.save_image(_str);
}

/**
* @brief compute the Hough accumulator in CUDA
* 
* @param[in] threshold, is the value to start counting a white pixel.
* @param[in] width, the width of the image to analyze.
* @param[in] height, the height of the image to analyze.
* @param[in] centerX, is width/2.
* @param[in] centerY, is height/2.
* @param[in] houghH, the maximum height depends on the image size.
* @param[in] pixels, array contanining the rgb values from the image to analyse.
* @param[out] accu, array contanining the accumulator score.
*/
__global__ void cudaAccumulator(const int _threshold, const int _width, const int _height, const double _centerX, const double _centerY, const double _houghH, const rgb_t* _pixels, int* _accu) {
    const auto row = threadIdx.y + blockIdx.y * blockDim.y;
    const auto col = threadIdx.x + blockIdx.x * blockDim.x;
    if (col >= _width || row >= _height) return;

    const auto index = row * _width + col;
    if ((_pixels[index].red + _pixels[index].green + _pixels[index].blue)/3 > _threshold) {
        for (auto t = 0; t < 180; ++t) {
            const auto r = (((double)col - _centerX) * cos((double)t * DEG2RAD)) + (((double)row - _centerY) * sin((double)t * DEG2RAD));
            const auto accuIndex = (int)((round(r + _houghH) * 180.0)) + t;
            atomicAdd(&_accu[accuIndex], 1);
        }
    }
}

/**
* @brief compute the Hough accumulator in CUDA
*
* @param[in] threshold, is the value to start counting a white pixel.
* @param[in] accuW, the width of the accumulator array.
* @param[in] accuH, the height of the accumulator array.
* @param[in] width, the width of the image to analyze.
* @param[in] height, the height of the image to analyze.
* @param[in] accu, array contanining the accumulator score.
*
* @param[out] lines, vector contanining the accumulator score.
*/
__global__ void cudaGetLines(const int _threshold, const int _accuW, const int _accuH, const int _width, const int _height, const int* _accu, line* _lines) {
    const int colT = threadIdx.x + blockIdx.x * blockDim.x;
    const int rowR = threadIdx.y + blockIdx.y * blockDim.y;
    if (colT >= _accuW || rowR >= _accuH) return;

    const int index = rowR * _accuW + colT;

    if (_accu[index] >= _threshold) {
        int max = _accu[index];
        for (int ly = -4; ly <= 4; ++ly)
            for (int lx = -4; lx <= 4; ++lx)
                if ((ly + rowR >= 0 && ly + rowR < _accuH) && (lx + colT >= 0 && lx + colT < _accuW))
                    if ((int)_accu[((rowR + ly) * _accuW) + (colT + lx)] > max)
                    {
                        max = _accu[((rowR + ly) * _accuW) + (colT + lx)];
                        ly = lx = 5;
                    }
        if (max > _accu[index] == false)
        {
            int x1, y1, x2, y2;

            if (colT >= 45 && colT <= 135)
            {
                //y = (r - x cos(t)) / sin(t)  
                x1 = 0;
                y1 = ((double)(rowR - (_accuH / 2)) - ((x1 - (_width / 2)) * cos(colT * DEG2RAD))) / sin(colT * DEG2RAD) + (_height / 2);
                x2 = _width - 0;
                y2 = ((double)(rowR - (_accuH / 2)) - ((x2 - (_width / 2)) * cos(colT * DEG2RAD))) / sin(colT * DEG2RAD) + (_height / 2);
            }
            else
            {
                //x = (r - y sin(t)) / cos(t);  
                y1 = 0;
                x1 = ((double)(rowR - (_accuH / 2)) - ((y1 - (_height / 2)) * sin(colT * DEG2RAD))) / cos(colT * DEG2RAD) + (_width / 2);
                y2 = _height - 0;
                x2 = ((double)(rowR - (_accuH / 2)) - ((y2 - (_height / 2)) * sin(colT * DEG2RAD))) / cos(colT * DEG2RAD) + (_width / 2);
            }

            _lines[index] = line{ x1, y1, x2, y2, true };
        }
    }
}